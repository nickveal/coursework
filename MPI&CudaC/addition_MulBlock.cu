#include "hip/hip_runtime.h"
#include <stdio.h>
//#include "../util.h"
//#include "hip/hip_runtime.h"
//#include "cutill_inline.h"
__global__ void matrixMul(float *A,float *B,float *C,int size,int width){
int bx=blockIdx.x;
int by=blockIdx.y;
int tx=threadIdx.x;
int ty=threadIdx.y;
int i;
int row=by*width+ty;
int column=bx*width+tx;
int temp;
temp=0;
for(i=0;i<size;i++){
	C[row*size+i]=A[row*size+i]+B[row*size+i];
}
}

int main(int argc , char **argv){
    int i,j;
    float* matrixA = NULL;
    float* matrixB = NULL;
   	float* matrixC = NULL;
   	int width;
   	int size,totalSize;
   	if(argc!=3){
		printf("format %s sizeofmatrix sizeofsmallmatrix\n",argv[0]);
		exit(1);	
	}
	size=atoi(argv[1]);
   	width=atoi(argv[2]);
   	printf("size=%d\n",size);
   	printf("width=%d\n",width);
   	totalSize= size*size;
   	printf("totalSize=%d\n",totalSize);
    float* data = (float*) malloc(totalSize*sizeof(float));
    (hipMalloc( (void**) &matrixA, totalSize*sizeof(float)));
    (hipMalloc( (void**) &matrixB, totalSize*sizeof(float)));
    (hipMalloc( (void**) &matrixC, totalSize*sizeof(float)));
	//checkErrors("Memory allocation");
    for(i = 0; i < totalSize; ++i) 
    {
        data[i] = 1;//(int)(10 * rand()/32768.f);
    }
    for(i=0;i<size;i++){
    	for(j=0;j<size;j++){
    	 //printf("%f ", data[i*size+j]);
    	}
    //printf("\n ");
    }
	printf("\n ");
    //checkErrors("Memory copy 0");
    (hipMemcpy( matrixA, data, sizeof(float)*totalSize, hipMemcpyHostToDevice) );
    (hipMemcpy( matrixB, data, sizeof(float)*totalSize, hipMemcpyHostToDevice) );
    //checkErrors("Memory copy 1");
    dim3 dimBlock(width,width);
    dim3 dimGrid(size/width,size/width);
	//
	hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event) ;
    hipEventCreate(&stop_event) ;
    hipEventRecord(start_event, 0);
	//
	matrixMul<<<dimGrid, dimBlock>>>(matrixA,matrixB,matrixC,size,width);
	hipDeviceSynchronize();
	//
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    float time_kernel; 
    hipEventElapsedTime(&time_kernel, start_event, stop_event);
	printf("Total time %f\n", time_kernel);
	//
	(hipMemcpy( data,matrixC, sizeof(float) * totalSize, hipMemcpyDeviceToHost));
    //checkErrors("Memory copy 2"); 

    for(i=0;i<size;i++){
    	for(j=0;j<size;j++){
    		//printf("%f ", data[i*size+j]);
    	}
    		//printf("\n ");
    } 
	free(data);
    hipFree( matrixA);
    hipFree( matrixB);
    hipFree( matrixC);
    return 0;
}
