#include "hip/hip_runtime.h"
#include <stdio.h>


__global__ void matrixMul(int *A,int *B,int *C,int size){
int tx=threadIdx.x;
int ty=threadIdx.y;
int i;
int temp;
temp=0;
for(i=0;i<size;i++){
	C[tx+ty*size]=A[tx+ty*size]+B[tx+ty*size];
}
printf("cuda");
}

int main(int argc , char **argv){
    int i,j;
    int* matrixA = NULL;
    int* matrixB = NULL;
   	int* matrixC = NULL;
   	int size,totalSize;
    if(argc!=2){
		printf("format:%s sizeofmatrix\n",argv[0]);
		exit(1);
	}
  	size=atoi(argv[1]);
   	totalSize= size*size;
    //allocate host memory
    int* data = (int*) malloc(totalSize*sizeof(int));
    //allocate device memory
    (hipMalloc( (void**) &matrixA, sizeof(int)*totalSize));
    (hipMalloc( (void**) &matrixB, sizeof(int)*totalSize));
    (hipMalloc( (void**) &matrixC, sizeof(int)*totalSize));
	
    for(i = 0; i < totalSize; i++) 
    {
        data[i] = 1;//(int)(10 * rand()/32768.f);
    }
    for(i=0;i<size;i++){
    	for(j=0;j<size;j++){
    	 printf("%d ", data[i*size+j]);
    	}
    printf("\n ");
    }
	printf("\n ");
    //copy data from memory to device memory
    (hipMemcpy( matrixA, data, sizeof(int)*totalSize, hipMemcpyHostToDevice) );
    (hipMemcpy( matrixB, data, sizeof(int)*totalSize, hipMemcpyHostToDevice) ); 
   
    dim3 dimBlock(size,size);
    dim3 dimGrid(1,1);
    //timing
	hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event) ;
    hipEventCreate(&stop_event) ;
    hipEventRecord(start_event, 0);
	//Call kernel(global function)
	matrixMul<<<dimGrid, dimBlock>>>(matrixA,matrixB,matrixC,size);
	hipDeviceSynchronize();
	//
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    float time_kernel; 
    hipEventElapsedTime(&time_kernel, start_event, stop_event);
	printf("Total time %f\n", time_kernel);
	//copy data from device memory to memory
	(hipMemcpy( data,matrixC, sizeof(int)*totalSize, hipMemcpyDeviceToHost));


    for(i=0;i<size;i++){
    	for(j=0;j<size;j++){
    	 	printf("%d ", data[i*size+j]);
    	}
    	printf("\n");
    } 
	free(data);
    hipFree( matrixA);
    hipFree( matrixB);
    hipFree( matrixC);
    return 0;
}
